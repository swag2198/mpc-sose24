// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009-2010
//
//   Ulm University
//
// Creator: Hendrik Lensch
// Email:   {hendrik.lensch,johannes.hanika}@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define MAX_BLOCKS 256
#define MAX_THREADS 128

//#define RTEST // use random initialization of array

/* compute the dot product between a1 and a2. a1 and a2 are of size
 dim. The result of each thread should be stored in _dst[blockIdx.x *
 blockDim.x + threadIdx.x]. Each thread should accumulate the dot
 product of a subset of elements.
 */
__global__ void dotProdKernel(float* _dst, const float* _a1, const float* _a2, int _dim)
{

    // program your kernel here
    //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int q = _dim / (MAX_BLOCKS * MAX_THREADS);
	int r = _dim % (MAX_BLOCKS * MAX_THREADS);
	int accumulate_size;

	if (q==0)
	{
		accumulate_size = 1;
	}
	else
	{
		accumulate_size = q;
		if (r > 0)
		{
			accumulate_size += 1;
		}
	}

	float temp = 0.0;
	int start_idx = idx * accumulate_size;  //each thread computes dot-product of accumulate_size arrays
	for(int k=0; k<accumulate_size; k++)
	{
		if (start_idx+k < _dim)
		{
			temp += _a1[start_idx+k] * _a2[start_idx+k];
		}
	}
	_dst[idx] = temp;

}

/* This program sets up two large arrays of size dim and computes the
dot product of both arrays.

The arrays are uploaded only once and the dot product is computed
multiple times. While this does not make too much sense it
demonstrated the possible speedup.  */
int main(int argc, char* argv[])
{
    // parse command line
    int acount = 1;

    if (argc < 3)
    {
        printf("usage: testDotProduct <dim> <GPU-flag [0,1]>\n");
        exit(1);
    }

    // number of elements in both vectors
    int dim = atoi(argv[acount++]);

    // flag indicating weather the CPU or the GPU version should be executed
    bool gpuVersion = atoi(argv[acount++]);

    printf("dim: %d\n", dim);

    float* cpuArray1 = new float[dim];
    float* cpuArray2 = new float[dim];

    // initialize the two arrays (either random or deterministic)
    for (int i = 0; i < dim; ++i)
    {
#ifdef RTEST
        cpuArray1[i] = drand48();
        cpuArray2[i] = drand48();
#else
        cpuArray1[i] = 2.0;
        //cpuArray2[i] = 1.0;
        cpuArray2[i] = i % 10;
#endif
    }

    // now the gpu stuff
    float* gpuArray1;
    float* gpuArray2;
    float* gpuResult;

    float* h;

    if (gpuVersion)
    {
        // allocate two gpuArray 1 and gpuArray 2 and gpuResult array on GPU
    	hipMalloc((void **)&gpuArray1, dim * sizeof(float));
    	hipMalloc((void **)&gpuArray2, dim * sizeof(float));
    	hipMalloc((void **)&gpuResult, MAX_BLOCKS * MAX_THREADS * sizeof(float));

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

        // copy the array once to the device
    	hipMemcpy(gpuArray1, cpuArray1, dim * sizeof(float), hipMemcpyHostToDevice);
    	hipMemcpy(gpuArray2, cpuArray2, dim * sizeof(float), hipMemcpyHostToDevice);

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

        // allocate an array to download the results of all threads
        h = new float[MAX_BLOCKS * MAX_THREADS];
    }

    const int num_iters = 100;
    double finalDotProduct = 0.0;

    if (!gpuVersion)
    {
        printf("cpu: ");
        for (int iter = 0; iter < num_iters; ++iter)
        {
            finalDotProduct = 0.0;
            for (int i = 0; i < dim; ++i)
            {
                finalDotProduct += cpuArray1[i] * cpuArray2[i];
            }
        }
    }
    else
    {

        // CUDA version here
        printf("gpu: ");

        // a simplistic way of splitting the problem into threads
        dim3 blockGrid(MAX_BLOCKS);
        dim3 threadBlock(MAX_THREADS);

        for (int iter = 0; iter < num_iters; ++iter)
        {
            dotProdKernel<<<blockGrid, threadBlock>>>(gpuResult, gpuArray1, gpuArray2, dim);
        }

        // download and combine the results of multiple threads on the CPU
        hipMemcpy(h, gpuResult, MAX_BLOCKS * MAX_THREADS * sizeof(float), hipMemcpyDeviceToHost);

        for (int i=0; i < MAX_BLOCKS * MAX_THREADS; i++)
        {
        	finalDotProduct += h[i];
        }

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

    }

    printf("Result: %f\n", finalDotProduct);

    if (gpuVersion)
    {

        // cleanup GPU memory

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!
    	hipFree(gpuArray1);
    	hipFree(gpuArray2);
    	hipFree(gpuResult);

        delete[] h;
    }

    delete[] cpuArray2;
    delete[] cpuArray1;

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
